#define CATCH_CONFIG_MAIN
#include "catch.hpp"

#include <string>
#include <iostream>
#include <hip/hip_runtime.h>

#include "test_utils.h"

#include "uint256_t.h"
#include "AES.h"

TEST_CASE( "uint256_t_eq_host", "[uint256_t]" )
{
    uint256_t an_int;
    uint256_t an_int2;

    bool eq = an_int == an_int2; 
    bool neq = an_int != an_int2;

    REQUIRE( eq );
    REQUIRE( !neq );

    uint256_t a1;
    uint256_t a2;

    for( std::uint8_t idx = 0; idx < UINT256_SIZE_IN_BYTES; ++idx )
        {
            a1[ idx ] = idx;
            a2[ idx ] = idx;
        }

    eq = a1 == a2;

    REQUIRE( eq );

    a1[ 0 ] = 0x01;
    a2[ 0 ] = 0x02;

    eq = a1 == a2;

    REQUIRE( !eq );
}

TEST_CASE( "uint256_t_eq_dev", "[uint256_t]" )
{
    uint256_t a1;
    uint256_t a2;

    bool *result_code_dev = nullptr;
    bool result_code = false;

    uint256_t *a1_dev = nullptr;
    uint256_t *a2_dev = nullptr;

    hipMalloc( (void**) &a1_dev, sizeof( uint256_t ) );
    hipMalloc( (void**) &a2_dev, sizeof( uint256_t ) );
    hipMalloc( (void**) &result_code_dev, sizeof( bool ) );

    if( test_utils::HtoD( a1_dev, &a1, sizeof( uint256_t ) ) != hipSuccess )
        {
            std::cout << "Failure to transfer a1 to device\n";
        }

    if( test_utils::HtoD( a2_dev, &a2, sizeof( uint256_t ) ) != hipSuccess)
        {
            std::cout << "Failure to transfer a2 to device\n";
        }
    if( test_utils::HtoD( result_code_dev, &result_code, sizeof( bool ) ) != hipSuccess)
        {
            std::cout << "Failure to transfer result_code to device\n";
        }

    test_utils::eq_kernel<<<1,1>>>( a1_dev,
                                    a2_dev,
                                    result_code_dev
                                  );

    if( test_utils::DtoH( &result_code, result_code_dev, sizeof( bool ) ) != hipSuccess)
        {
            std::cout << "Failure to transfer to host \n";
        }

    REQUIRE( result_code );

    result_code = false;

    a1[ 0 ] = 0x02;

    if( test_utils::HtoD( a1_dev, &a1, sizeof( uint256_t ) ) != hipSuccess )
        {
            std::cout << "Failure to transfer a1 to device\n";
        }

    if( test_utils::HtoD( result_code_dev, &result_code, sizeof( bool ) ) != hipSuccess)
        {
            std::cout << "Failure to transfer result_code to device\n";
        }

    test_utils::eq_kernel<<<1,1>>>( a1_dev,
                                    a2_dev,
                                    result_code_dev
                                  );

    if( test_utils::DtoH( &result_code, result_code_dev, sizeof( bool ) ) != hipSuccess)
        {
            std::cout << "Failure to transfer to host \n";
        }

    REQUIRE( !result_code );


    hipFree( a1_dev );
    hipFree( a2_dev );
    hipFree( result_code_dev );

}