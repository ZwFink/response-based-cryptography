#define CATCH_CONFIG_MAIN
#include "catch.hpp"

#include <string>
#include <iostream>
#include <hip/hip_runtime.h>
#include <algorithm>
#include <cstdlib>

#include "test_utils.h"

#include "aes_per_round.h"
#include "uint256_iterator.h"
#include "uint256_t.h"
#include "AES.h"

TEST_CASE( "uint256_t_eq_host", "[uint256_t]" )
{
    uint256_t an_int;
    uint256_t an_int2;

    bool eq = an_int == an_int2; 
    bool neq = an_int != an_int2;

    REQUIRE( eq );
    REQUIRE( !neq );

    uint256_t a1;
    uint256_t a2;

    for( std::uint8_t idx = 0; idx < UINT256_SIZE_IN_BYTES; ++idx )
        {
            a1[ idx ] = idx;
            a2[ idx ] = idx;
        }

    eq = a1 == a2;

    REQUIRE( eq );

    a1[ 0 ] = 0x01;
    a2[ 0 ] = 0x02;

    eq = a1 == a2;

    REQUIRE( !eq );
}

TEST_CASE( "uint256_t_eq_dev", "[uint256_t]" )
{
    uint256_t a1;
    uint256_t a2;

    bool *result_code_dev = nullptr;
    bool result_code = false;

    uint256_t *a1_dev = nullptr;
    uint256_t *a2_dev = nullptr;

    hipMalloc( (void**) &a1_dev, sizeof( uint256_t ) );
    hipMalloc( (void**) &a2_dev, sizeof( uint256_t ) );
    hipMalloc( (void**) &result_code_dev, sizeof( bool ) );

    if( test_utils::HtoD( a1_dev, &a1, sizeof( uint256_t ) ) != hipSuccess )
        {
            std::cout << "Failure to transfer a1 to device\n";
        }

    if( test_utils::HtoD( a2_dev, &a2, sizeof( uint256_t ) ) != hipSuccess)
        {
            std::cout << "Failure to transfer a2 to device\n";
        }
    if( test_utils::HtoD( result_code_dev, &result_code, sizeof( bool ) ) != hipSuccess)
        {
            std::cout << "Failure to transfer result_code to device\n";
        }

    test_utils::binary_op_kernel<uint256_t, &uint256_t::operator==><<<1,1>>>( a1_dev, a2_dev, result_code_dev );

    if( test_utils::DtoH( &result_code, result_code_dev, sizeof( bool ) ) != hipSuccess)
        {
            std::cout << "Failure to transfer to host \n";
        }

    REQUIRE( result_code );

    result_code = false;

    a1[ 0 ] = 0x02;

    if( test_utils::HtoD( a1_dev, &a1, sizeof( uint256_t ) ) != hipSuccess )
        {
            std::cout << "Failure to transfer a1 to device\n";
        }

    if( test_utils::HtoD( result_code_dev, &result_code, sizeof( bool ) ) != hipSuccess)
        {
            std::cout << "Failure to transfer result_code to device\n";
        }

    test_utils::binary_op_kernel<uint256_t, &uint256_t::operator==><<<1,1>>>( a1_dev, a2_dev, result_code_dev );

    if( test_utils::DtoH( &result_code, result_code_dev, sizeof( bool ) ) != hipSuccess)
        {
            std::cout << "Failure to transfer to host \n";
        }

    REQUIRE( !result_code );

    hipFree( a1_dev );
    hipFree( a2_dev );
    hipFree( result_code_dev );

}

TEST_CASE( "uint256_t_negation_cpu", "[uint256_t]" )
{

    uint256_t a1;
    uint256_t a2;

    bool cond = false;

    auto check_reqs = [&]()
        {
            cond = a1 == ~a2;
            REQUIRE( cond );

            cond = ~a1 == a2;
            REQUIRE( cond );

        };


    for( std::uint8_t idx = 0; idx < UINT256_SIZE_IN_BYTES; ++idx )
        {
            a1[ idx ] = 0xFF; // a1 == ~a2
            a2[ idx ] = 0x00;
        }

    check_reqs();

    for( std::uint8_t idx = 0; idx < UINT256_SIZE_IN_BYTES; idx += 2 )
        {
            a1[ idx ] = 0x00; // a1 == ~a2
            a2[ idx ] = 0xFF;
        }

    check_reqs();

    for( std::uint8_t idx = 0; idx < UINT256_SIZE_IN_BYTES; idx += 2 )
        {
            a1[ idx ] = ( rand() % 256 );
            a2[ idx ] = ~(a1[idx]);
        }

    check_reqs();
}

TEST_CASE( "uint256_t_negation_gpu", "[uint256_t]" )
{

    uint256_t a1;
    uint256_t a2;

    uint256_t *a1_dev = nullptr;
    uint256_t *a2_dev = nullptr;

    hipMalloc( (void**) &a1_dev, sizeof( uint256_t ) );
    hipMalloc( (void**) &a2_dev, sizeof( uint256_t ) );

    bool result_code = false;

    if( test_utils::HtoD( a1_dev, &a1, sizeof( uint256_t ) ) != hipSuccess )
        {
            std::cout << "Failure to transfer a1 to device\n";
        }

    if( test_utils::HtoD( a2_dev, &a2, sizeof( uint256_t ) ) != hipSuccess)
        {
            std::cout << "Failure to transfer a2 to device\n";
        }

    test_utils::unary_op_kernel<uint256_t, &uint256_t::operator~><<<1,1>>>
        ( a1_dev,
          a2_dev
        );

    if( test_utils::DtoH( &a2, a2_dev, sizeof( uint256_t ) ) != hipSuccess)
        {
            std::cout << "Failure to transfer to host \n";
        }

    result_code = a2 == ~a1;

    REQUIRE( result_code );

    hipFree( a1_dev );
    hipFree( a2_dev );
}

TEST_CASE( "uint256_t_ctz_popc", "[uint256_t]" )
{
    uint256_t my_int;
    uint256_t my_int_2;
    int z_count = 0;

    uint256_t *my_int_dev;
    uint256_t *my_int_dev_2;
    int *z_count_dev;

    hipMalloc( (void**) &my_int_dev, sizeof( uint256_t ) );
    hipMalloc( (void**) &my_int_dev_2, sizeof( uint256_t ) );
    hipMalloc( (void**) &z_count_dev, sizeof( int ) );

    bool result_code = false;

    if( test_utils::HtoD( my_int_dev, &my_int, sizeof( uint256_t ) ) != hipSuccess )
        {
            std::cout << "Failure to transfer a1 to device\n";
        }

    if( test_utils::HtoD( z_count_dev, &z_count, sizeof( int ) ) != hipSuccess)
        {
            std::cout << "Failure to transfer a2 to device\n";
        }

    test_utils::popc<<<1,1>>>( my_int_dev, z_count_dev );

    if( test_utils::DtoH( &z_count, z_count_dev, sizeof( int ) ) != hipSuccess)
        {
            std::cout << "Failure to transfer a2 to device\n";
        }

    REQUIRE( z_count == 0 );

    test_utils::ctz<<<1,1>>>( my_int_dev, z_count_dev );

    if( test_utils::DtoH( &z_count, z_count_dev, sizeof( int ) ) != hipSuccess)
        {
            std::cout << "Failure to transfer a2 to device\n";
        }

    REQUIRE( z_count == 256 );

    test_utils::unary_op_kernel<uint256_t, &uint256_t::operator~><<<1,1>>>
        ( my_int_dev,
          my_int_dev_2
        );

    test_utils::popc<<<1,1>>>( my_int_dev_2, z_count_dev );

    if( test_utils::DtoH( &z_count, z_count_dev, sizeof( int ) ) != hipSuccess)
        {
            std::cout << "Failure to transfer a2 to device\n";
        }

    REQUIRE( z_count == 256 );

    test_utils::ctz<<<1,1>>>( my_int_dev_2, z_count_dev );

    if( test_utils::DtoH( &z_count, z_count_dev, sizeof( int ) ) != hipSuccess)
        {
            std::cout << "Failure to transfer a2 to device\n";
        }

    REQUIRE( z_count == 0 );

    hipFree( my_int_dev );
    hipFree( my_int_dev_2 );
    hipFree( z_count_dev );
}

TEST_CASE( "uint256_t_<<", "[uint256_t]" )
{

    uint256_t large = UINT256_MAX_INT;

    uint256_t shifted = large << 256;

    bool success = shifted == UINT256_ZERO;

    REQUIRE( success );

    // reset it
    large.set_all( 0xFF );

    // check first bytes = 16 = 0xFF, last 16 = 0x00
    shifted = large << 128;


    for( int x = 0; x < UINT256_SIZE_IN_BYTES / 2; ++x )
        {
            REQUIRE( shifted[ x ] == 0x00 );
        }
    for( int x = UINT256_SIZE_IN_BYTES / 2; x < UINT256_SIZE_IN_BYTES; ++x )
        {
            REQUIRE( shifted[ x ] == 0xFF );
        }

    large.set_all( 0xFF );
    shifted = large << 2;

    for( int x = 31; x > 0; --x )
        {
            REQUIRE( shifted[ x ] == 0xFF );
        }

    REQUIRE( shifted[ 0 ] == 0xFC );

    large.set_all( 0xFF );

    shifted = large << 1;

    for( int x = 31; x > 0; --x )
        {
            REQUIRE( shifted[ x ] == 0xFF );
        }

    REQUIRE( shifted[ 0 ] == 0xFE );
}

TEST_CASE( "uint256_t_>>", "[uint256_t]" )
{

    uint256_t large = UINT256_MAX_INT;

    uint256_t shifted = large >> 256;

    bool success = shifted == UINT256_ZERO;

    REQUIRE( success );

    // reset it
    large.set_all( 0xFF );

    // check first bytes = 16 = 0xFF, last 16 = 0x00
    shifted = large >> 128;


    for( int x = 0; x < UINT256_SIZE_IN_BYTES / 2; ++x )
        {
            REQUIRE( shifted[ x ] == 0xFF );
        }
    for( int x = UINT256_SIZE_IN_BYTES / 2; x < UINT256_SIZE_IN_BYTES; ++x )
        {
            REQUIRE( shifted[ x ] == 0x00 );
        }

    large.set_all( 0xFF );
    shifted = large >> 2;

    for( int x = 0; x < 31; ++x )
        {
            REQUIRE( shifted[ x ] == 0xFF );
        }

    REQUIRE( shifted[ 31 ] == 0x3F );

    large.set_all( 0xFF );

    shifted = large >> 1;

    for( int x = 0; x < 31; ++x )
        {
            REQUIRE( shifted[ x ] == 0xFF );
        }

    REQUIRE( shifted[ 31 ] == 0x7F );
}

TEST_CASE( "uint256_t_add", "[uint256_t]" )
{
    uint256_t a1;
    uint256_t a2;
    uint256_t result;

    uint256_t *a1_dev = nullptr;
    uint256_t *a2_dev = nullptr;
    uint256_t *result_dev = nullptr;

    hipMalloc( (void**) &a1_dev, sizeof( uint256_t ) );
    hipMalloc( (void**) &a2_dev, sizeof( uint256_t ) );
    hipMalloc( (void**) &result_dev, sizeof( uint256_t ) );

    bool result_code = false;

    SECTION( "0+255=0" )
        {
            a1.set_all( 0xFF );
            a2.set_all( 0x00 );
            result.set_all( 0x00 );

            if( test_utils::HtoD( a1_dev, &a1, sizeof( uint256_t ) ) != hipSuccess )
                {
                    std::cout << "Failure to transfer a1 to device\n";
                }

            if( test_utils::HtoD( a2_dev, &a2, sizeof( uint256_t ) ) != hipSuccess)
                {
                    std::cout << "Failure to transfer a2 to device\n";
                }

            if( test_utils::HtoD( result_dev, &result, sizeof( uint256_t ) ) != hipSuccess)
                {
                    std::cout << "Failure to transfer result to device\n";
                }

            test_utils::add_knl<<<1,1>>>( a1_dev, a2_dev, result_dev );
            hipDeviceSynchronize();

            if( test_utils::DtoH( &result, result_dev, sizeof( uint256_t ) ) != hipSuccess)
                {
                    std::cout << "Failure to transfer to host \n";
                }

            result_code = result == UINT256_MAX_INT;
            REQUIRE( result_code );
        }

    SECTION( "188+67=255" )
        {
            // 188 + 67 = 255

            // 188
            a1.set_all( 0xBC );

            // 67
            a2.set_all( 0x43 );

            if( test_utils::HtoD( a1_dev, &a1, sizeof( uint256_t ) ) != hipSuccess )
                {
                    std::cout << "Failure to transfer a1 to device\n";
                }

            if( test_utils::HtoD( a2_dev, &a2, sizeof( uint256_t ) ) != hipSuccess)
                {
                    std::cout << "Failure to transfer a2 to device\n";
                }

            if( test_utils::HtoD( result_dev, &result, sizeof( uint256_t ) ) != hipSuccess)
                {
                    std::cout << "Failure to transfer result to device\n";
                }

            test_utils::add_knl<<<1,1>>>( a1_dev, a2_dev, result_dev );
            hipDeviceSynchronize();

            if( test_utils::DtoH( &result, result_dev, sizeof( uint256_t ) ) != hipSuccess)
                {
                    std::cout << "Failure to transfer to host \n";
                }

            result_code = result == UINT256_MAX_INT;
            REQUIRE( result_code );
        }

    hipFree( a1_dev );
    hipFree( a2_dev );

}

TEST_CASE( "uint256_t<", "[uint256_t]" )
{
    uint256_t a1( 0x00 );
    uint256_t a2( 0x00 );
    bool result = false;

    SECTION( "0<255" )
        {
            a1.set_all( 0xFF );

            result = a2 < a1;

            REQUIRE( a2 < a1 );
        }
    SECTION( "n-1<n" )
        {
            a1.set_all( 0x43 );
            a2.set_all( 0x43 );

            a2[ 0 ] = 0x42;
            result = a2 < a1;

            REQUIRE( result );
        }
    SECTION( "n < x, x way bigger than n" )
        {
            a1.set_all( 0x43 );
            a2.set_all( 0x43 );
            a1[ 31 ] = 0x44;

            result = a2 < a1;

            REQUIRE( result );


        }
    SECTION( "!(n < n)" )
        {
            a1.set_all( 0xFF );
            a2.set_all( 0xFF );

            result = a1 < a2;

            REQUIRE( !result );

            result = a2 < a1;

            REQUIRE( !result );
        }

}

TEST_CASE( "uint256_t>", "[uint256_t]" )
{
    uint256_t a1( 0x00 );
    uint256_t a2( 0x00 );
    bool result = false;

    SECTION( "255>0" )
        {
            a2.set_all( 0xFF );

            result = a2 > a1;

            REQUIRE( result );
        }
    SECTION( "n>n-n" )
        {
            a1.set_all( 0x43 );
            a2.set_all( 0x43 );

            a2[ 0 ] = 0x44;
            result = a2 > a1;

            REQUIRE( result );
        }
    SECTION( "x > n, x way bigger than n" )
        {
            a1.set_all( 0x43 );
            a2.set_all( 0x43 );
            a2[ 31 ] = 0x44;

            result = a2 > a1;

            REQUIRE( result );


        }
    SECTION( "!(n > n)" )
        {
            a1.set_all( 0xFF );
            a2.set_all( 0xFF );

            result = a1 > a2;

            REQUIRE( !result );

            result = a2 > a1;

            REQUIRE( !result );
        }

}

TEST_CASE( "uint256_t::neg", "[uint256_t]" )
{
    uint256_t a( 0xFF );
    uint256_t b( 0x01, UINT256_SIZE_IN_BYTES - 1 );
    uint256_t c( 0x00 );

    uint256_t *a_dev = nullptr;
    uint256_t *c_dev = nullptr;

    hipMalloc( (void**) &a_dev, sizeof( uint256_t ) );
    hipMalloc( (void**) &c_dev, sizeof( uint256_t ) );

    if( test_utils::HtoD( a_dev, &a, sizeof( uint256_t ) ) != hipSuccess )
        {
            std::cout << "Failure to transfer a to device\n";
        }
    if( test_utils::HtoD( c_dev, &c, sizeof( uint256_t ) ) != hipSuccess )
        {
            std::cout << "Failure to transfer c to device\n";
        }

    test_utils::neg_knl<<<1,1>>>( a_dev, c_dev );


    if( test_utils::DtoH( &c, c_dev, sizeof( uint256_t ) ) != hipSuccess )
        {
            std::cout << "Failure to transfer c to device\n";
        }

    bool result = c == b;
    REQUIRE( result );
}


TEST_CASE( "uint256_iter_constructor", "[uint256_iterator]" )
{

    uint256_t a( 0x00 );
    uint256_t b( 0xFF );
    uint256_iter( (const unsigned char *)
                  "abcdefghijklmnopqrstuvwxyz012345",
                  a,
                  b
                );

}