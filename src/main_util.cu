#include "hip/hip_runtime.h"
// utility file for host driver funciton (main)

#include "main_util.h"
#include <stdio.h>

__global__ void kernel_rbc_engine( uint256_t *key_for_encryp,
                                   uint256_t *key_to_find,
                                   const int mismatch,
                                   const aes_per_round::message_128 *user_id,
                                   const aes_per_round::message_128 *auth_cipher,
                                   const std::size_t key_sz_bits,
                                   const std::size_t num_blocks,
                                   const std::size_t threads_per_block,
                                   const std::size_t keys_per_thread,
                                   std::uint64_t *iter_count
                                 )
{
    unsigned int tid = threadIdx.x + ( blockIdx.x * blockDim.x );

    uint256_t starting_perm, ending_perm;

    uint64_t num_keys = 0;
    int result        = 0;

    num_keys = get_bin_coef( key_sz_bits, mismatch ); 
   
    // only run thread if tid is less than cardinality of current keyspace
    if( tid < num_keys )
    {
        get_perm_pair( &starting_perm, 
                       &ending_perm, 
                       (std::size_t) tid, 
                       (std::size_t) num_blocks * threads_per_block,
                       mismatch,
                       keys_per_thread,
                       key_sz_bits
                     );
        
        result = validator( &starting_perm,
                            &ending_perm,
                            key_for_encryp,
                            user_id,
                            auth_cipher
                          );

        // if result is 1 then we found a key matching client's private key
        // signal all threads to stop
        // if( result )
        //     {
        //         *key_to_find = *key_for_encryp; 
        //     }
        atomicAdd( (unsigned long long int*) iter_count, result );
    }

}

__device__ int validator( uint256_t *starting_perm,
                          uint256_t *ending_perm,
                          uint256_t *key_for_encryp,
                          const aes_per_round::message_128 *user_id,
                          const aes_per_round::message_128 *auth_cipher 
                        )
{
    aes_per_round::message_128 encrypted;
    int idx = 0;
    std::uint8_t match = 0;
    std::uint8_t match2 = 0;
    int total = 0;

    for( idx = 0; idx < 4; ++idx )
        {
            ((uint32_t*)&(encrypted.bits))[ idx ] = 0;
        }

    #ifdef USE_SMEM
    __shared__ std::uint8_t sbox[ SBOX_SIZE_IN_BYTES ];
    if( threadIdx.x < SBOX_SIZE_IN_BYTES )
        {
            sbox[ threadIdx.x ] = Tsbox_256[ threadIdx.x ];
        }

    __syncthreads();

    #else
    // just get a reference to it
    uint8_t *sbox = Tsbox_256;
    #endif 

    uint256_iter iter ( *key_for_encryp,
                        *starting_perm,
                        *ending_perm
                      );
    while( !iter.end() )
        {

            ++total;
            // encrypt
            aes_per_round::roundwise_encrypt( &encrypted,
                                              &iter.corrupted_key,
                                              user_id,
                                              sbox
                                            );

            // check for match! 
            for( idx = 0; idx < 16; ++idx )
                {
                    match += ( encrypted.bits[ idx ] == auth_cipher->bits[ idx ] );
                }
            match2 += match == 16; // if all 16 bytes matched, we have a match!

            if( match == 16 )
                {
                    *key_for_encryp = iter.corrupted_key;
                    printf( "I found it!\n" );
                }

            match = 0;

            // get next key
            iter.next();

            for( idx = 0; idx < 4; ++idx )
                {
                    ((uint32_t*)&(encrypted.bits))[ idx ] = 0;
                }

        }
    return total;
    // return match2;
}

void warm_up_gpu( int device )
{
    hipSetDevice( device ); 		
    // initialize all ten integers of a device_vector to 1 
    thrust::device_vector<int> D(10, 1); 
    // set the first seven elements of a vector to 9 
    thrust::fill(D.begin(), D.begin() + 7, 9); 
    // initialize a host_vector with the first five elements of D 
    thrust::host_vector<int> H(D.begin(), D.begin() + 5); 
    // set the elements of H to 0, 1, 2, 3, ... 
    thrust::sequence(H.begin(), H.end()); // copy all of H back to the beginning of D 
    thrust::copy(H.begin(), H.end(), D.begin()); 
    // print D 

    printf("\nDevice: %d\n",device);

    for(int i = 0; i < D.size(); i++) 
        std::cout << " D[" << i << "] = " << D[i]; 


    // empty the vector
    D.clear();

    // deallocate any capacity which may currently be associated with vec
    D.shrink_to_fit();

    printf("\n");

    return;
}


