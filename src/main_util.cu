#include "hip/hip_runtime.h"
// utility file for host driver funciton (main)

#include "main_util.h"
#include "aes_tables.h"
#include "util.h"
#include <stdio.h>

__global__ void kernel_rbc_engine( uint256_t *key_for_encryp,
                                   uint256_t *key_to_find,
                                   const int mismatch,
                                   const aes_per_round::message_128 *user_id,
                                   const aes_per_round::message_128 *auth_cipher,
                                   const std::size_t key_sz_bits,
                                   const std::size_t num_blocks,
                                   const std::size_t threads_per_block,
                                   const std::size_t keys_per_thread,
                                   std::uint64_t num_keys,
                                   std::uint16_t extra_keys,
                                   std::uint64_t *iter_count
                                 )
{
    unsigned int tid = threadIdx.x + ( blockIdx.x * blockDim.x );

    uint256_t starting_perm, ending_perm;

    int result = 0;

    // only run thread if tid is less than cardinality of current keyspace
    if( tid < num_keys )
    {
        get_perm_pair( &starting_perm, 
                       &ending_perm, 
                       (uint64_t) tid, 
                       (uint64_t) num_blocks * threads_per_block,
                       mismatch,
                       keys_per_thread,
                       extra_keys
                     );
        
        result = validator( &starting_perm,
                            &ending_perm,
                            key_for_encryp,
                            key_to_find,
                            user_id,
                            auth_cipher
                          );

        atomicAdd( (unsigned long long int*) iter_count, result );
    }

}

__host__ __device__ uint bytes_to_int( const std::uint8_t *bytes )
{
    uint ret_val;
    ret_val =  ((int)bytes[3] << 24) | ((int)bytes[2] << 16) | ((int)bytes[1] << 8) | ((int)bytes[0]);
    return ret_val;
}
__device__ int validator( uint256_t *starting_perm,
                          uint256_t *ending_perm,
                          uint256_t *key_for_encryp,
                          uint256_t *key_to_find,
                          const aes_per_round::message_128 *user_id,
                          const aes_per_round::message_128 *auth_cipher 
                        )
{
    // aes_per_round::message_128 encrypted;

    uint userid[ 4 ];
    uint cyphertext[ 4 ];
    uint authcipher[ 4 ];

    userid[ 0 ] = bytes_to_int( user_id->bits );
    userid[ 1 ] = bytes_to_int( user_id->bits + 4 );
    userid[ 2 ] = bytes_to_int( user_id->bits + 8 );
    userid[ 3 ] = bytes_to_int( user_id->bits + 12 );

    authcipher[ 0 ] = bytes_to_int( auth_cipher->bits );
    authcipher[ 1 ] = bytes_to_int( auth_cipher->bits + 4 );
    authcipher[ 2 ] = bytes_to_int( auth_cipher->bits + 8 );
    authcipher[ 3 ] = bytes_to_int( auth_cipher->bits + 12 );

    aes_tables tabs;
    std::uint8_t idx = 0;
    std::uint8_t match = 0;
    int total = 0;

    #ifdef USE_SMEM
    __shared__ std::uint8_t sbox[ SBOX_SIZE_IN_BYTES ];
    if( threadIdx.x < SBOX_SIZE_IN_BYTES )
        {
            #if THREADS_PER_BLOCK == 128 

            sbox[ 2 * threadIdx.x ] = Tsbox_256[ 2 * threadIdx.x ];
            sbox[ ( 2 * threadIdx.x ) + 1 ] = Tsbox_256[ ( 2 * threadIdx.x ) + 1 ];

            #elif THREADS_PER_BLOCK == 256

            sbox[ threadIdx.x ] = Tsbox_256[ threadIdx.x ];

            #endif
        }
    __shared__ uint Te0[256], Te1[256], Te2[256], Te3[256];
    load_smem(Te0, cTe0, Te1, cTe1, Te2, cTe2, Te3, cTe3);
    // NOTE: __syncthreads not used here because it's called in
    // util::load_smem

    tabs.Te0 = Te0;
    tabs.Te1 = Te1;
    tabs.Te2 = Te2;
    tabs.Te3 = Te3;

    #else
    // just get a reference to it
    uint8_t *sbox = Tsbox_256;

    tabs.Te0 = cTe0;
    tabs.Te1 = cTe1;
    tabs.Te2 = cTe2;
    tabs.Te3 = cTe3;

    #endif 

    tabs.sbox = sbox;

    uint256_iter iter ( *key_for_encryp,
                        *starting_perm,
                        *ending_perm
                      );

    while( !iter.end() )
        {

            ++total;
            // encrypt
            aes_gpu::encrypt( userid,
                              cyphertext,
                              (uint*)(iter.corrupted_key.data),
                              &tabs
                            );

            // check for match! 
            for( idx = 0; idx < 4; ++idx )
                {
                    match += ( cyphertext[ idx ] == authcipher[ idx ] );
                }

           if( match == 4 )
                {
                    *key_to_find = iter.corrupted_key;
                    printf( "I found it!\n" );
                }

            match = 0;

            // get next key
            iter.next();

        }
    return total;
}

void warm_up_gpu( int device )
{
    hipSetDevice( device ); 		
    // initialize all ten integers of a device_vector to 1 
    thrust::device_vector<int> D(10, 1); 
    // set the first seven elements of a vector to 9 
    thrust::fill(D.begin(), D.begin() + 7, 9); 
    // initialize a host_vector with the first five elements of D 
    thrust::host_vector<int> H(D.begin(), D.begin() + 5); 
    // set the elements of H to 0, 1, 2, 3, ... 
    thrust::sequence(H.begin(), H.end()); // copy all of H back to the beginning of D 
    thrust::copy(H.begin(), H.end(), D.begin()); 
    // print D 

    printf("\nDevice: %d\n",device);

    for(int i = 0; i < D.size(); i++) 
        std::cout << " D[" << i << "] = " << D[i]; 


    // empty the vector
    D.clear();

    // deallocate any capacity which may currently be associated with vec
    D.shrink_to_fit();

    printf("\n");

    return;
}


