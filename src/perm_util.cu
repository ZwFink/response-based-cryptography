#include "hip/hip_runtime.h"
// utility file for permutation delimination

#ifndef PERM_UTIL_CU_
#define PERM_UTIL_CU_

#include "uint256_t.h"

// COMPLETED
__device__ void decode_ordinal( uint256_t perm, 
                                const uint256_t ordinal, 
                                size_t mismatches, // 0-6
                                int key_sz_bits    // 256
                              )
{
   uint256_t binom, wkg_ordinal;
   uint64_t tmp_binom    = 0;
   uint64_t tmp_curr_ord = 0;
   wkg_ordinal.copy( ordinal );
   perm.set_all( 0 );

   for( size_t bit = key_sz_bits-1; mismatches > 0; bit-- )
   {
      tmp_binom = get_bin_coef( bit, mismatches );
      binom( 0 );
      binom( tmp_binom, 2 );

      if ( wkg_ordinal > binom || wkg_ordinal == binom )
      {
         wkg_ordinal = wkg_ordinal - binom;

         perm.set_bit( bit );

         mismatches--;
      }
   }
}

// COMPLETED
__device__ void assign_first_permutation( uint256_t *perm, int mismatches )
{
   // set perm to first key

   *perm = *perm << mismatches; // shift left
	
   perm->add( *perm, UINT256_NEGATIVE_ONE ); // add negative one
}

// COMPLETED
__device__ void assign_last_permutation( uint256_t *perm,
                                         int mismatches,
                                         int key_sz_bits )
{

   // set perm to the first key
   assign_first_permutation( perm, mismatches );

   // Then find the last key by shifting the first
   // Equiv to: perm << (key_length - mismatches)
   // E.g. if key_length = 256 and mismatches = 5,
   //      we want to shift left 256 - 5 = 251 times.
   *perm = *perm << (key_sz_bits - mismatches);
}

// COMPLETED
// Precondition: starting_perm and ending_perm have been initialized
__device__ void get_perm_pair( uint256_t *starting_perm, 
                               uint256_t *ending_perm,
                               size_t pair_index,        // thread num
                               size_t pair_count,        // num threads
                               int mismatches,           // 5
                               size_t key_size_bytes,    // 32  (key_size)
                               size_t key_sz_bits        // 256 (key_sz_bits)
                             )
{
   uint256_t total_perms();
   uint256_t starting_ordinal();
   uint256_t ending_ordinal();
   uint64_t tmp_tot_perms; 
   uint64_t tmp_starting_ord; 
   uint64_t tmp_ending_ord; 

   tmp_tot_perms = get_bin_coef( key_sz_bits, mismatches );
   total_perms( tmp_tot_perms, 2 ); 

   if( pair_index == 0 )
   {
      assign_first_permutation( starting_perm, mismatches );
   } 
   else
   {
      tmp_starting_ord = floor( tmp_tot_perms / pair_count );
      tmp_starting_ord = tmp_starting_ord * pair_index;
      // copy 64 bit tmp into uint256_t at index 2 
      // uint256_t is big endian - most significant byte first
      starting_ordinal( tmp_starting_ord, 2 );

      decode_ordinal(starting_perm, starting_ordinal, mismatches, key_sz_bits);
   }

   if( pair_index == pair_count - 1 )
   {
      assign_last_permutation( ending_perm, mismatches, key_sz_bits );
   } 
   else
   {
      tmp_ending_ord = floor( tmp_tot_perms / pair_count );
      tmp_starting_ord = tmp_ending_ord * (pair_index + 1);
      starting_ordinal( tmp_starting_ord, 2 ); // copy into uint256_t
   
      decode_ordinal(ending_perm, ending_ordinal, mismatches, key_sz_bits);
   }
}

// compute the binomial coefficient:
// get the number of k-element subsets of an n-element set
__device__ uint64_t get_bin_coef(size_t n, size_t r)
{
   int i;
   uint64_t b;

   if( (r < 0) || (n < r) ) 
      return 0;

   if( (2*r) > n ) 
      r = n-r;

   b = 1;

   if( r>0 )
   {
      for( i=0; i<=r-1; i++ )
      {
         b = ( b*(n-i) ) / (i+1);
      }
   }

   return b;
}

// we don't need this here -- should be used in main before kernel invocation.
__device__ void get_random_permutation( uint256_t perm,
                                        int mismatches,
                                        int key_sz_bits )
{


}

// we don't need this here -- should be used in main before kernel invocation.
__device__ void get_benchmark_permutation( uint256_t perm,
                                           int mismatches,
                                           int key_sz_bits )
{


}
   
#endif

                             
