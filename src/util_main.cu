#include "hip/hip_runtime.h"
// utility file for host driver funciton (main)

#include "util_main.h"
#include "aes_tables.h"
#include "util.h"
#include <stdio.h>

__global__ void kernel_rbc_engine( uint256_t *key_for_encryp,
                                   uint256_t *key_to_find,
                                   const int mismatch,
                                   const uint *user_id,
                                   const uint *auth_cipher,
                                   const std::size_t num_blocks,
                                   const std::size_t threads_per_block,
                                   const std::size_t keys_per_thread,
                                   const std::uint64_t num_keys,
                                   const std::uint32_t extra_keys,
                                   std::uint64_t *iter_count,
                                   int *key_found_flag,
                                   const uint64_t offset,
                                   const int gpu_id
                                   //const int CHECKCOUNT
                                 )
{
    unsigned int tid = threadIdx.x + ( blockIdx.x * blockDim.x ) + ( gpu_id * offset );

    aes_tables tabs;

    #ifdef USE_SMEM
    __shared__ std::uint8_t sbox[ SBOX_SIZE_IN_BYTES ];
    if( threadIdx.x < SBOX_SIZE_IN_BYTES )
        {
            #if THREADS_PER_BLOCK == 128 

            sbox[ 2 * threadIdx.x ] = Tsbox_256[ 2 * threadIdx.x ];
            sbox[ ( 2 * threadIdx.x ) + 1 ] = Tsbox_256[ ( 2 * threadIdx.x ) + 1 ];

            #elif THREADS_PER_BLOCK == 256

            sbox[ threadIdx.x ] = Tsbox_256[ threadIdx.x ];

            #endif
        }
    __shared__ uint Te0[256], Te1[256], Te2[256], Te3[256];
    load_smem(Te0, cTe0, Te1, cTe1, Te2, cTe2, Te3, cTe3);
    // NOTE: __syncthreads not used here because it's called in
    // util::load_smem

    tabs.Te0 = Te0;
    tabs.Te1 = Te1;
    tabs.Te2 = Te2;
    tabs.Te3 = Te3;

    #else
    // just get a reference to it
    uint8_t *sbox = Tsbox_256;

    tabs.Te0 = cTe0;
    tabs.Te1 = cTe1;
    tabs.Te2 = cTe2;
    tabs.Te3 = cTe3;

    #endif 

    if( tid < (gpu_id+1)*offset )
    {
        uint256_t starting_perm, ending_perm;
        tabs.sbox = sbox;
        std::uint8_t idx = 0;
        std::uint8_t match = 0;
        int total = 0;
        uint cyphertext[ 4 ];

        get_perm_pair( &starting_perm, 
                       &ending_perm, 
                       (uint64_t) tid, 
                       (uint64_t) num_blocks * threads_per_block,
                       mismatch,
                       keys_per_thread,
                       extra_keys
                     );

        uint256_iter iter ( *key_for_encryp,
                            starting_perm,
                            ending_perm
                          );

        while( !iter.end() )
            {

                ++total;
                // encrypt
                aes_gpu::encrypt( user_id,
                                  cyphertext,
                                  (uint*)(iter.corrupted_key.data),
                                  &tabs
                                );

                // check for match! 
                for( idx = 0; idx < 4; ++idx )
                    {
                        match += ( cyphertext[ idx ] == auth_cipher[ idx ] );
                    }

                if( match == 4 )
                    {
                        *key_to_find = iter.corrupted_key;
                        if( EARLY_EXIT )
                            atomicAdd( (unsigned long long int*) key_found_flag, 1 );
                        //__trap();
                    }

                match = 0;

                // get next key
                iter.next();

                // exit early strategy
                if( EARLY_EXIT && (total%ITERCOUNT)==0 && *key_found_flag )
                    break;

            }

        atomicAdd( (unsigned long long int*) iter_count, total );
    }

}

__host__ __device__ uint bytes_to_int( const std::uint8_t *bytes )
{
    uint ret_val;
    ret_val =  ((int)bytes[3] << 24) | ((int)bytes[2] << 16) | ((int)bytes[1] << 8) | ((int)bytes[0]);
    return ret_val;
}

void warm_up_gpu( int device, int verbose )
{
    hipSetDevice( device ); 		
    // initialize all ten integers of a device_vector to 1 
    thrust::device_vector<int> D(10, 1); 
    // set the first seven elements of a vector to 9 
    thrust::fill(D.begin(), D.begin() + 7, 9); 
    // initialize a host_vector with the first five elements of D 
    thrust::host_vector<int> H(D.begin(), D.begin() + 5); 
    // set the elements of H to 0, 1, 2, 3, ... 
    thrust::sequence(H.begin(), H.end()); // copy all of H back to the beginning of D 
    thrust::copy(H.begin(), H.end(), D.begin()); 
    // print D 

    if( verbose ) printf("\nDevice: %d\n",device);

    for(int i = 0; i < D.size(); i++) 
        if( verbose ) std::cout << " D[" << i << "] = " << D[i]; 


    // empty the vector
    D.clear();

    // deallocate any capacity which may currently be associated with vec
    D.shrink_to_fit();

    if( verbose ) printf("\n");

    return;
}


