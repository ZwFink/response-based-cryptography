#include "hip/hip_runtime.h"
#include <iostream>
#include <iomanip>

#include "uint256_t.h"

uint256_t::uint256_t()
{
    set_all( 0 );
}


CUDA_CALLABLE_MEMBER uint256_t::uint256_t( std::uint8_t set, std::uint8_t index )
{
    data[ index ] = set;
}

CUDA_CALLABLE_MEMBER uint256_t::uint256_t( std::uint8_t set )
{
    set_all( set );
}

CUDA_CALLABLE_MEMBER void uint256_t::set_all( std::uint8_t val )
{
    memset( data, val, UINT256_SIZE_IN_BYTES );
}

CUDA_CALLABLE_MEMBER std::uint8_t& uint256_t::operator[]( std::uint8_t idx )
{
    return data[ idx ];
}

CUDA_CALLABLE_MEMBER uint256_t uint256_t::operator~()
{
    uint256_t ret;

    for( std::uint8_t index = 0;
         index < UINT256_SIZE_IN_BYTES;
         ++index
       )
        {
            ret[ index ] = ~data[ index ];
        }
    return ret;
}

CUDA_CALLABLE_MEMBER uint256_t uint256_t::operator&( uint256_t comp )
{
    uint256_t ret;

    for( std::uint8_t index = 0;
         index < UINT256_SIZE_IN_BYTES;
         ++index
       )
        {
            ret[ index ] = comp[ index ] & data[ index ];
        }

    return ret;
}

CUDA_CALLABLE_MEMBER uint256_t uint256_t::operator^( uint256_t comp )
{
    uint256_t ret;

    for( std::uint8_t index = 0;
         index < UINT256_SIZE_IN_BYTES;
         ++index
       )
        {
            ret[ index ] = comp[ index ] ^ data[ index ];
        }

    return ret;
}

CUDA_CALLABLE_MEMBER uint256_t uint256_t::operator|( uint256_t comp )
{
    uint256_t ret;

    for( std::uint8_t index = 0;
         index < UINT256_SIZE_IN_BYTES;
         ++index
       )
        {
            ret[ index ] = comp[ index ] | data[ index ];
        }

    return ret;
}

CUDA_CALLABLE_MEMBER bool uint256_t::operator==( uint256_t comp )
{
    bool ret = true;
    for( uint8_t byte = 0; byte < UINT256_SIZE_IN_BYTES; ++byte )
        {
            ret = ret && ( data[ byte ] == comp[ byte ] );
        }
    return ret;
}

CUDA_CALLABLE_MEMBER uint256_data_t& uint256_t::get_data()
{
    return data;
}

CUDA_CALLABLE_MEMBER bool uint256_t::operator!=( uint256_t comp )
{
    return !( *this == comp );
}

__host__ void uint256_t::dump()
{
    for( const auto& x : data )
        {
            std::cout
                << "0x"
                << std::setfill('0')
                << std::setw(2)
                << std::hex
                << unsigned( x )
                << " ";
        }
    std::cout << "\n"; 
}

CUDA_CALLABLE_MEMBER uint256_t uint256_t::operator>>( int shift )
{
    uint256_t ret;

    std::uint8_t limb_shifts  = shift / UINT256_LIMB_SIZE;
    std::uint8_t shift_length = shift % UINT256_LIMB_SIZE;

    std::uint8_t byte = 0;

    for( byte = limb_shifts; byte < UINT256_SIZE_IN_BYTES; ++byte )
        {
            ret[ byte - limb_shifts ] = data[ byte ];
        }

    // leading limbs are alread zero

    for( byte = 0; byte < UINT256_SIZE_IN_BYTES - 1; ++byte )
        {
            ret[ byte ] = ( ret.at( byte ) >> shift_length
                            | ( ret.at( byte + 1 ) << ( UINT256_LIMB_SIZE - shift_length ) )
                          );
        }

    ret[ UINT256_SIZE_IN_BYTES - 1 ] >>= shift;

    return ret;
}

CUDA_CALLABLE_MEMBER uint256_t uint256_t::operator<<( int shift )
{
    uint256_t ret;

    std::uint8_t limb_shifts  = shift / UINT256_LIMB_SIZE;
    std::uint8_t shift_length = shift % UINT256_LIMB_SIZE;

    std::uint8_t byte = 0;

    for( byte = 0; byte < UINT256_SIZE_IN_BYTES - limb_shifts; ++byte )
        {
            ret[ byte + limb_shifts ] = data[ byte ];
        }

    // trailing limbs are alread zero

    for( byte = UINT256_SIZE_IN_BYTES - 1; byte > 0; --byte )
        {
            ret[ byte ] = ret.at( byte ) << shift_length
                | ( ret.at( byte - 1 ) >> ( UINT256_LIMB_SIZE - shift_length ) );
        }
    ret[ 0 ] <<= shift_length;
    
    return ret;

}

__device__ int uint256_t::popc()
{
    int total_ones = 0;
    std::uint32_t *current = nullptr;

    for( std::uint8_t index = 0; index < UINT256_SIZE_IN_BYTES / 4; ++index )
        {
            current = (std::uint32_t*) data + index;
            total_ones += __popc( *current );
        }

    return total_ones;
}

CUDA_CALLABLE_MEMBER std::uint8_t uint256_t::at( int loc )
{
    return data[ loc ];
}

__device__ int uint256_t::ctz()
{
    return 256 - popc();
}

CUDA_CALLABLE_MEMBER void uint256_t::to_32_bit_arr( std::uint32_t* dest )
{
    memcpy( dest, &(data), 32 );
}

CUDA_CALLABLE_MEMBER int uint256_t::compare( const uint256_t& comp ) const
{
    std::uint32_t *my_data = (std::uint32_t*) &data;
    std::uint32_t *comp_data = (std::uint32_t*) &comp.data;

    int result = 0;

    for( int index = ( UINT256_SIZE_IN_BYTES / 4 ) - 1;
         !result && index >= 0;
         --index
       )
        {
            result = ( my_data[ index ] > comp_data[ index ] )
                     - ( my_data[ index ] < comp_data[ index ] );
        }
    return result;
}

CUDA_CALLABLE_MEMBER bool uint256_t::operator<( const uint256_t& comp ) const
{
    return compare( comp ) < 0;
}

CUDA_CALLABLE_MEMBER bool uint256_t::operator>( const uint256_t& comp ) const
{
    return compare( comp ) > 0;
}


__device__ uint256_t uint256_t::add( uint256_t augend )
{
    uint256_t ret;

    std::uint32_t *self_32   = (uint32_t*) &data;
    std::uint32_t *augend_32 = (uint32_t*) &augend.data;
    std::uint32_t *ret_32    = (uint32_t*) &ret.data;

    asm ("add.cc.u32      %0, %8, %16;\n\t"
         "addc.cc.u32     %1, %9, %17;\n\t"
         "addc.cc.u32     %2, %10, %18;\n\t"
         "addc.cc.u32     %3, %11, %19;\n\t"
         "addc.cc.u32     %4, %12, %20;\n\t"
         "addc.cc.u32     %5, %13, %21;\n\t"
         "addc.cc.u32     %6, %14, %22;\n\t"
         "addc.u32        %7, %15, %23;\n\t"
         : "=r"(ret_32[ 0 ]), "=r"(ret_32[ 1 ]), "=r"(ret_32[ 2 ]),   
           "=r"(ret_32[ 3 ]), "=r"(ret_32[ 4 ]), "=r"(ret_32[ 5 ]),   
           "=r"(ret_32[ 6 ]), "=r"(ret_32[ 7 ])
         : "r"(self_32[ 0 ]), "r"(self_32[ 1 ]), "r"(self_32[ 2 ]),   
           "r"(self_32[ 3 ]), "r"(self_32[ 4 ]), "r"(self_32[ 5 ]),   
           "r"(self_32[ 6 ]), "r"(self_32[ 7 ]),
           "r"(augend_32[ 0 ]), "r"(augend_32[ 1 ]), "r"(augend_32[ 2 ]),   
           "r"(augend_32[ 3 ]), "r"(augend_32[ 4 ]), "r"(augend_32[ 5 ]),   
           "r"(augend_32[ 6 ]), "r"(augend_32[ 7 ])
         );

    return ret;
}

__device__ void uint256_t::neg( uint256_t& dest )
{
    uint256_t complement = ~(*this);
    uint256_t one( 0x00 );
    one[ 0 ] = 0x01;

    dest = complement.add( one );
}