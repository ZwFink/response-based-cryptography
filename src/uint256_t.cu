#include "hip/hip_runtime.h"
#include <iostream>
#include <iomanip>

#include "uint256_t.h"

uint256_t::uint256_t()
{
    set_all( 0 );
}

CUDA_CALLABLE_MEMBER uint256_t::uint256_t( std::uint8_t set )
{
    set_all( set );
}

CUDA_CALLABLE_MEMBER void uint256_t::set_all( std::uint8_t val )
{
    memset( data, val, UINT256_SIZE_IN_BYTES );
}

CUDA_CALLABLE_MEMBER std::uint8_t& uint256_t::operator[]( std::uint8_t idx )
{
    return data[ idx ];
}

CUDA_CALLABLE_MEMBER uint256_t uint256_t::operator~()
{
    uint256_t ret;

    for( std::uint8_t index = 0;
         index < UINT256_SIZE_IN_BYTES;
         ++index
       )
        {
            ret[ index ] = ~data[ index ];
        }
    return ret;
}

CUDA_CALLABLE_MEMBER uint256_t uint256_t::operator&( uint256_t comp )
{
    uint256_t ret;

    for( std::uint8_t index = 0;
         index < UINT256_SIZE_IN_BYTES;
         ++index
       )
        {
            ret[ index ] = comp[ index ] & data[ index ];
        }

    return ret;
}

CUDA_CALLABLE_MEMBER uint256_t uint256_t::operator^( uint256_t comp )
{
    uint256_t ret;

    for( std::uint8_t index = 0;
         index < UINT256_SIZE_IN_BYTES;
         ++index
       )
        {
            ret[ index ] = comp[ index ] ^ data[ index ];
        }

    return ret;
}

CUDA_CALLABLE_MEMBER uint256_t uint256_t::operator|( uint256_t comp )
{
    uint256_t ret;

    for( std::uint8_t index = 0;
         index < UINT256_SIZE_IN_BYTES;
         ++index
       )
        {
            ret[ index ] = comp[ index ] | data[ index ];
        }

    return ret;
}

CUDA_CALLABLE_MEMBER bool uint256_t::operator==( uint256_t comp )
{
    bool ret = true;
    for( uint8_t byte = 0; byte < UINT256_SIZE_IN_BYTES; ++byte )
        {
            ret = ret && ( data[ byte ] == comp[ byte ] );
        }
    return ret;
}

CUDA_CALLABLE_MEMBER uint256_data_t& uint256_t::get_data()
{
    return data;
}

CUDA_CALLABLE_MEMBER bool uint256_t::operator!=( uint256_t comp )
{
    return !( *this == comp );
}

__host__ void uint256_t::dump()
{
    for( const auto& x : data )
        {
            std::cout
                << "0x"
                << std::setfill('0')
                << std::setw(2)
                << std::hex
                << unsigned( x )
                << " ";
        }
    std::cout << "\n"; 
}

CUDA_CALLABLE_MEMBER uint256_t uint256_t::operator>>( int shift )
{
    uint256_t ret;

    std::uint8_t limb_shifts  = shift / UINT256_LIMB_SIZE;
    std::uint8_t shift_length = shift % UINT256_LIMB_SIZE;

    std::uint8_t byte = 0;

    for( byte = limb_shifts; byte < UINT256_SIZE_IN_BYTES; ++byte )
        {
            ret[ byte - limb_shifts ] = data[ byte ];
        }

    // leading limbs are alread zero

    for( byte = 0; byte < UINT256_SIZE_IN_BYTES - 1; ++byte )
        {
            ret[ byte ] = ( ret.at( byte ) >> shift_length
                            | ( ret.at( byte + 1 ) << ( UINT256_LIMB_SIZE - shift_length ) )
                          );
        }

    ret[ UINT256_SIZE_IN_BYTES - 1 ] >>= shift;

    return ret;
}

CUDA_CALLABLE_MEMBER uint256_t uint256_t::operator<<( int shift )
{
    uint256_t ret;

    std::uint8_t limb_shifts  = shift / UINT256_LIMB_SIZE;
    std::uint8_t shift_length = shift % UINT256_LIMB_SIZE;

    std::uint8_t byte = 0;

    for( byte = 0; byte < UINT256_SIZE_IN_BYTES - limb_shifts; ++byte )
        {
            ret[ byte + limb_shifts ] = data[ byte ];
        }

    // trailing limbs are alread zero

    for( byte = UINT256_SIZE_IN_BYTES - 1; byte > 0; --byte )
        {
            ret[ byte ] = ret.at( byte ) << shift_length
                | ( ret.at( byte - 1 ) >> ( UINT256_LIMB_SIZE - shift_length ) );
        }
    ret[ 0 ] <<= shift_length;
    
    return ret;

}

__device__ int uint256_t::popc()
{
    int total_ones = 0;
    uint32_t current = 0;

    for( std::uint8_t index = 0; index < 32; index += 4 )
        {
            current |= data[ index ];
            current = current << 8;

            current |= data[ index + 1 ];
            current = current << 8;

            current |= data[ index + 2 ];
            current = current << 8;

            current |= data[ index + 3 ];

            total_ones += __popc( current );
            current = 0;
        }

    return total_ones;
}

CUDA_CALLABLE_MEMBER std::uint8_t uint256_t::at( int loc )
{
    return data[ loc ];
}

__device__ int uint256_t::ctz()
{
    return 256 - popc();
}

CUDA_CALLABLE_MEMBER void uint256_t::to_32_bit_arr( std::uint32_t* dest )
{
    memcpy( dest, &(data), 32 );
}

CUDA_CALLABLE_MEMBER uint256_t uint256_t::add( uint256_t augend )
{
    uint256_t ret;

    uint256_t_casted *self_32   = (uint256_t_casted*) ((void*) &data );
    uint256_t_casted *augend_32 = (uint256_t_casted*) ((void*) &augend.data );
    uint256_t_casted *ret_32    = (uint256_t_casted*) ((void*) &ret.data );

    asm ("add.cc.u32      %0, %8, %16;\n\t"
         "addc.cc.u32     %1, %9, %17;\n\t"
         "addc.cc.u32     %2, %10, %18;\n\t"
         "addc.cc.u32     %3, %11, %19;\n\t"
         "addc.cc.u32     %4, %12, %20;\n\t"
         "addc.cc.u32     %5, %13, %21;\n\t"
         "addc.cc.u32     %6, %14, %22;\n\t"
         "addc.u32        %7, %15, %23;\n\t"
         : "=r"(ret_32->a), "=r"(ret_32->b), "=r"(ret_32->c),   
           "=r"(ret_32->d), "=r"(ret_32->e), "=r"(ret_32->f),   
           "=r"(ret_32->g), "=r"(ret_32->h)
         : "r"(self_32->a), "r"(self_32->b), "r"(self_32->c),   
           "r"(self_32->d), "r"(self_32->e), "r"(self_32->f),   
           "r"(self_32->g), "r"(self_32->h),
           "r"(augend_32->a), "r"(augend_32->b), "r"(augend_32->c),   
           "r"(augend_32->d), "r"(augend_32->e), "r"(augend_32->f),   
           "r"(augend_32->g), "r"(augend_32->h)
       );

    return ret;
}