#include "hip/hip_runtime.h"
//#define TTABLE 256
//#define USE_SMEM
#include "util.cu"
#include "tabs/sbox.tab"
union u32_t {
    uint i;
    uchar c[4];
};

#if TTABLE == 256
#include "tabs/AES_256.tab"
#define TE(tab, offset, state) (_te[(tab)][((offset) << 8) + (state)])

#elif TTABLE == 128
#include "tabs/AES_128.tab"
#define TE_128_LH(tab, offset, state) (_te[(tab)][((offset) << 8) + (state >> 1)      ] >> ((state & 0x1) << 2))
#define TE_128_HH(tab, offset, state) (_te[(tab)][((offset) << 8) + (state >> 1) + 128] >> ((state & 0x1) << 2))
#define TE(tab, offset, state) ((TE_128_LH(tab, offset, state) & 0x0f) | (TE_128_HH(tab, offset, state) << 4))

#elif TTABLE == 64
#include "tabs/AES_64.tab"
#define TE_64_0(tab, offset, state) (_te[(tab)][((offset) << 8) + (state >> 2)      ] >> ((state & 0x3) << 1))
#define TE_64_1(tab, offset, state) (_te[(tab)][((offset) << 8) + (state >> 2) +  64] >> ((state & 0x3) << 1))
#define TE_64_2(tab, offset, state) (_te[(tab)][((offset) << 8) + (state >> 2) + 128] >> ((state & 0x3) << 1))
#define TE_64_3(tab, offset, state) (_te[(tab)][((offset) << 8) + (state >> 2) + 192] >> ((state & 0x3) << 1))
#define TE(tab, offset, state) ((TE_64_0(tab, offset, state) & 0x03)	\
				| ((TE_64_1(tab, offset, state) & 0x03) << 2) \
				| ((TE_64_2(tab, offset, state) & 0x03) << 4) \
				| ((TE_64_3(tab, offset, state) & 0x03) << 6))

#elif TTABLE == 32
#include "tabs/AES_32.tab"
#define TE_32_0(tab, offset, state) (_te[(tab)][((offset) << 8) + (state >> 3)      ] >> (state & 0x7))
#define TE_32_1(tab, offset, state) (_te[(tab)][((offset) << 8) + (state >> 3) +  32] >> (state & 0x7))
#define TE_32_2(tab, offset, state) (_te[(tab)][((offset) << 8) + (state >> 3) +  64] >> (state & 0x7))
#define TE_32_3(tab, offset, state) (_te[(tab)][((offset) << 8) + (state >> 3) +  96] >> (state & 0x7))
#define TE_32_4(tab, offset, state) (_te[(tab)][((offset) << 8) + (state >> 3) + 128] >> (state & 0x7))
#define TE_32_5(tab, offset, state) (_te[(tab)][((offset) << 8) + (state >> 3) + 160] >> (state & 0x7))
#define TE_32_6(tab, offset, state) (_te[(tab)][((offset) << 8) + (state >> 3) + 192] >> (state & 0x7))
#define TE_32_7(tab, offset, state) (_te[(tab)][((offset) << 8) + (state >> 3) + 224] >> (state & 0x7))
#define TE(tab, offset, state) ((TE_32_0(tab, offset, state) & 0x01)	      \
				| ((TE_32_1(tab, offset, state) & 0x01) << 1) \
				| ((TE_32_2(tab, offset, state) & 0x01) << 2) \
				| ((TE_32_3(tab, offset, state) & 0x01) << 3) \
				| ((TE_32_4(tab, offset, state) & 0x01) << 4) \
				| ((TE_32_5(tab, offset, state) & 0x01) << 5) \
				| ((TE_32_6(tab, offset, state) & 0x01) << 6) \
				| ((TE_32_7(tab, offset, state) & 0x01) << 7))		   
#endif


#define LOAD_U8_G0(s, offset) {					\
	m[0].c[offset] = TE(0, offset, s[0].c[3]);			\
	m[1].c[offset] = TE(1, offset, s[1].c[2]);			\
	m[2].c[offset] = TE(2, offset, s[2].c[1]);			\
	m[3].c[offset] = TE(3, offset, s[3].c[0]);			\
    }

#define LOAD_U8_G1(s, offset) {						\
	m[0].c[offset] = TE(0, offset, s[1].c[3]);			\
	m[1].c[offset] = TE(1, offset, s[2].c[2]);			\
	m[2].c[offset] = TE(2, offset, s[3].c[1]);			\
	m[3].c[offset] = TE(3, offset, s[0].c[0]);			\
    }
#define LOAD_U8_G2(s, offset) {						\
	m[0].c[offset] = TE(0, offset, s[2].c[3]);			\
	m[1].c[offset] = TE(1, offset, s[3].c[2]);			\
	m[2].c[offset] = TE(2, offset, s[0].c[1]);			\
	m[3].c[offset] = TE(3, offset, s[1].c[0]);			\
    }
#define LOAD_U8_G3(s, offset) {						\
	m[0].c[offset] = TE(0, offset, s[3].c[3]);			\
	m[1].c[offset] = TE(1, offset, s[0].c[2]);			\
	m[2].c[offset] = TE(2, offset, s[1].c[1]);			\
	m[3].c[offset] = TE(3, offset, s[2].c[0]);			\
    }

#define SECURE_ROUND(t, s, r) {						\
	int b = r << 2;							\
	u32_t m[4];							\
	LOAD_U8_G0(s, 0); LOAD_U8_G0(s, 1); LOAD_U8_G0(s, 2); LOAD_U8_G0(s, 3); \
	t[0].i = m[0].i ^ m[1].i ^ m[2].i ^ m[3].i ^ _rk[b + 0];	\
	LOAD_U8_G1(s, 0); LOAD_U8_G1(s, 1); LOAD_U8_G1(s, 2); LOAD_U8_G1(s, 3);	\
	t[1].i = m[0].i ^ m[1].i ^ m[2].i ^ m[3].i ^ _rk[b + 1];	\
	LOAD_U8_G2(s, 0); LOAD_U8_G2(s, 1); LOAD_U8_G2(s, 2); LOAD_U8_G2(s, 3);	\
	t[2].i = m[0].i ^ m[1].i ^ m[2].i ^ m[3].i ^ _rk[b + 2];	\
	LOAD_U8_G3(s, 0); LOAD_U8_G3(s, 1); LOAD_U8_G3(s, 2); LOAD_U8_G3(s, 3);	\
	t[3].i = m[0].i ^ m[1].i ^ m[2].i ^ m[3].i ^ _rk[b + 3];	\
    }


#if TTABLE == 128
#define STE_128_LH(state) (sbox[(state >> 1)      ] >> ((state & 0x1) << 2))
#define STE_128_HH(state) (sbox[(state >> 1) + 128] >> ((state & 0x1) << 2))
#define STE(state) ((STE_128_LH(state) & 0x0f) | (STE_128_HH(state) << 4))
#elif TTABLE == 64
#define STE_64_0(state) (sbox[(state >> 2)      ] >> ((state & 0x3) << 1))
#define STE_64_1(state) (sbox[(state >> 2) +  64] >> ((state & 0x3) << 1))
#define STE_64_2(state) (sbox[(state >> 2) + 128] >> ((state & 0x3) << 1))
#define STE_64_3(state) (sbox[(state >> 2) + 192] >> ((state & 0x3) << 1))
#define STE(state)                 ((STE_64_0(state) & 0x03)	 \
				| ((STE_64_1(state) & 0x03) << 2) \
				| ((STE_64_2(state) & 0x03) << 4) \
				| ((STE_64_3(state) & 0x03) << 6))
#elif TTABLE == 32
#define STE_32_0(state) (sbox[(state >> 3)      ] >> (state & 0x7))
#define STE_32_1(state) (sbox[(state >> 3) +  32] >> (state & 0x7))
#define STE_32_2(state) (sbox[(state >> 3) +  64] >> (state & 0x7))
#define STE_32_3(state) (sbox[(state >> 3) +  96] >> (state & 0x7))
#define STE_32_4(state) (sbox[(state >> 3) + 128] >> (state & 0x7))
#define STE_32_5(state) (sbox[(state >> 3) + 160] >> (state & 0x7))
#define STE_32_6(state) (sbox[(state >> 3) + 192] >> (state & 0x7))
#define STE_32_7(state) (sbox[(state >> 3) + 224] >> (state & 0x7))
#define STE(state) ((STE_32_0(state) & 0x01)			\
		 | ((STE_32_1(state) & 0x01) << 1)			\
		 | ((STE_32_2(state) & 0x01) << 2)			\
		 | ((STE_32_3(state) & 0x01) << 3)			\
		 | ((STE_32_4(state) & 0x01) << 4)			\
		 | ((STE_32_5(state) & 0x01) << 5)			\
		 | ((STE_32_6(state) & 0x01) << 6)			\
		 | ((STE_32_7(state) & 0x01) << 7))		   
#else
#define STE(state) (sbox[state])
#endif

#define SWAP(a, b) (a) ^= (b); (b) ^= (a); (a) ^= (b);
__device__ void TransposeSelf(uchar *state) {
    SWAP(state[1], state[4]);
    SWAP(state[2], state[8]);
    SWAP(state[3], state[12]);
    SWAP(state[6], state[9]);
    SWAP(state[7], state[13]);
    SWAP(state[11], state[14]);
}

__device__ void Transpose(uchar *dst, uchar *src) {
    for (int i = 0; i < 4; i++) {
	for (int j = 0; j < 4; j++) {
	    dst[j*4+i] = src[i*4+j];
	}
    }
}
__device__ void AddRoundKey(uchar *state, uchar *rek) {
    for (int i = 0; i < 4; i++) {
	for (int j = 0; j < 4; j++) {
	    state[j*4+i] ^= rek[i*4+3-j];
	}
    }
}
__device__ void SubBytes(uchar *state, uchar *sbox) {
    for (int i = 0; i < 16; i++) {
	state[i] = STE(state[i]);//sbox[state[i]];
	//state[i] = Tsbox_256[state[i]];
    }
}

#define xtime(x)   ((x << 1) ^ ((x >> 7) * 0x1b))
__device__ void MixColumns(uchar *state) {
    uchar Tmp, Tm, t;
    for(int i = 0; i < 4; i++) {
	t                  = state[i];
	Tmp                = state[i] ^ state[4+i] ^ state[8+i] ^ state[12+i] ;

	Tm                 = state[i] ^ state[4+i] ; 
	Tm                 = xtime(Tm); 
	state[i] ^= Tm ^ Tmp ;

	Tm                 = state[4+i] ^ state[8+i] ; 
	Tm                 = xtime(Tm); 
	state[4+i] ^= Tm ^ Tmp ;

	Tm                 = state[8+i] ^ state[12+i] ; 
	Tm                 = xtime(Tm); 
	state[8+i] ^= Tm ^ Tmp ;

	Tm                 = state[12+i] ^ t ; 
	Tm                 = xtime(Tm); 
	state[12+i] ^= Tm ^ Tmp ;
    }
}

__device__ void ShiftRows(uchar *state) {
    uchar temp;
    // Rotate first row 1 columns to left    
    temp     = state[4];
    state[4] = state[5];
    state[5] = state[6];
    state[6] = state[7];
    state[7] = temp;
    
    // Rotate second row 2 columns to left    
    temp        = state[8];
    state[8] = state[10];
    state[10] = temp;
    
    temp       = state[9];
    state[9] = state[11];
    state[11] = temp;
    
    // Rotate third row 3 columns to left
    temp              = state[12];
    state[12] = state[15];
    state[15] = state[14];
    state[14] = state[13];
    state[13] = temp;
}

#define REV_ENDIAN(x) (((x)>>24)&0x000000FF) | (((x)>>8)&0x0000FF00) | (((x)<<8)&0x00FF0000) | (((x)<<24)&0xFF000000)


__global__ void AES_encrypt(const uint *pt, uint *ct, uint *rek, uint Nr, uint size) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    u32_t state[4];
//    uchar state[16];
    u32_t _s[4], _t[4];
    uint s0, s1, s2, s3, t0, t1, t2, t3;
#ifdef USE_SMEM
    __shared__ uchar sbox[256];
    __shared__ uchar sTe0[1024], sTe1[1024], sTe2[1024], sTe3[1024];
    uchar* _te[] = {sTe0, sTe1, sTe2, sTe3};
    __shared__ uint Te0[256], Te1[256], Te2[256], Te3[256];
    load_smem(Te0, cTe0, Te1, cTe1, Te2, cTe2, Te3, cTe3);
#if TTABLE == 256
    load_smem_sbox(sbox, Tsbox_256);
    load_smem(sTe0, dTe0_256, sTe1, dTe1_256, sTe2, dTe2_256, sTe3, dTe3_256);
#elif TTABLE == 128
    load_smem_sbox(sbox, Tsbox_128);
    load_smem(sTe0, dTe0_128, sTe1, dTe1_128, sTe2, dTe2_128, sTe3, dTe3_128);
#elif TTABLE == 64
    load_smem_sbox(sbox, Tsbox_64);
    load_smem(sTe0, dTe0_64, sTe1, dTe1_64, sTe2, dTe2_64, sTe3, dTe3_64);
#elif TTABLE == 32
    load_smem_sbox(sbox, Tsbox_32);
    load_smem(sTe0, dTe0_32, sTe1, dTe1_32, sTe2, dTe2_32, sTe3, dTe3_32);
#endif // TTABLE
#else
    uint *Te0 = cTe0, *Te1 = cTe1, *Te2 = cTe2, *Te3 = cTe3;
#if TTABLE == 256
    uchar *sbox = Tsbox_256;
    uchar* _te[] = {dTe0_256, dTe1_256, dTe2_256, dTe3_256};
#elif TTABLE == 128
    uchar *sbox = Tsbox_128;
    uchar* _te[] = {dTe0_128, dTe1_128, dTe2_128, dTe3_128};
#elif TTABLE == 64
    uchar *sbox = Tsbox_64;
    uchar* _te[] = {dTe0_64, dTe1_64, dTe2_64, dTe3_64};
#elif TTABLE == 32
    uchar *sbox = Tsbox_32;
    uchar* _te[] = {dTe0_32, dTe1_32, dTe2_32, dTe3_32};
#endif // TTABLE
#endif // USE_SMEM

    int iter = 0;

 BEGIN:
    int offset = (iter * NUM_THREADS * NUM_BLOCKS + tid) << 2;
    if (offset >= size) return;

    uint *_rk = rek;
    /*
     * map byte array block to cipher state
     * and add initial round key:
     */
    s0 = pt[offset + 0] ^ _rk[0];
    s1 = pt[offset + 1] ^ _rk[1];
    s2 = pt[offset + 2] ^ _rk[2];
    s3 = pt[offset + 3] ^ _rk[3];

    /* round 1: */
    t0 = Te0[s0 >> 24] ^ Te1[(s1 >> 16) & 0xff] ^ Te2[(s2 >>  8) & 0xff] ^ Te3[s3 & 0xff] ^ _rk[ 4];
    t1 = Te0[s1 >> 24] ^ Te1[(s2 >> 16) & 0xff] ^ Te2[(s3 >>  8) & 0xff] ^ Te3[s0 & 0xff] ^ _rk[ 5];
    t2 = Te0[s2 >> 24] ^ Te1[(s3 >> 16) & 0xff] ^ Te2[(s0 >>  8) & 0xff] ^ Te3[s1 & 0xff] ^ _rk[ 6];
    t3 = Te0[s3 >> 24] ^ Te1[(s0 >> 16) & 0xff] ^ Te2[(s1 >>  8) & 0xff] ^ Te3[s2 & 0xff] ^ _rk[ 7];
    /* round 2: */
    s0 = Te0[t0 >> 24] ^ Te1[(t1 >> 16) & 0xff] ^ Te2[(t2 >>  8) & 0xff] ^ Te3[t3 & 0xff] ^ _rk[ 8];
    s1 = Te0[t1 >> 24] ^ Te1[(t2 >> 16) & 0xff] ^ Te2[(t3 >>  8) & 0xff] ^ Te3[t0 & 0xff] ^ _rk[ 9];
    s2 = Te0[t2 >> 24] ^ Te1[(t3 >> 16) & 0xff] ^ Te2[(t0 >>  8) & 0xff] ^ Te3[t1 & 0xff] ^ _rk[10];
    s3 = Te0[t3 >> 24] ^ Te1[(t0 >> 16) & 0xff] ^ Te2[(t1 >>  8) & 0xff] ^ Te3[t2 & 0xff] ^ _rk[11];
    /* round 3: */
    t0 = Te0[s0 >> 24] ^ Te1[(s1 >> 16) & 0xff] ^ Te2[(s2 >>  8) & 0xff] ^ Te3[s3 & 0xff] ^ _rk[12];
    t1 = Te0[s1 >> 24] ^ Te1[(s2 >> 16) & 0xff] ^ Te2[(s3 >>  8) & 0xff] ^ Te3[s0 & 0xff] ^ _rk[13];
    t2 = Te0[s2 >> 24] ^ Te1[(s3 >> 16) & 0xff] ^ Te2[(s0 >>  8) & 0xff] ^ Te3[s1 & 0xff] ^ _rk[14];
    t3 = Te0[s3 >> 24] ^ Te1[(s0 >> 16) & 0xff] ^ Te2[(s1 >>  8) & 0xff] ^ Te3[s2 & 0xff] ^ _rk[15];
    /* round 4: */
    s0 = Te0[t0 >> 24] ^ Te1[(t1 >> 16) & 0xff] ^ Te2[(t2 >>  8) & 0xff] ^ Te3[t3 & 0xff] ^ _rk[16];
    s1 = Te0[t1 >> 24] ^ Te1[(t2 >> 16) & 0xff] ^ Te2[(t3 >>  8) & 0xff] ^ Te3[t0 & 0xff] ^ _rk[17];
    s2 = Te0[t2 >> 24] ^ Te1[(t3 >> 16) & 0xff] ^ Te2[(t0 >>  8) & 0xff] ^ Te3[t1 & 0xff] ^ _rk[18];
    s3 = Te0[t3 >> 24] ^ Te1[(t0 >> 16) & 0xff] ^ Te2[(t1 >>  8) & 0xff] ^ Te3[t2 & 0xff] ^ _rk[19];
    /* round 5: */
    t0 = Te0[s0 >> 24] ^ Te1[(s1 >> 16) & 0xff] ^ Te2[(s2 >>  8) & 0xff] ^ Te3[s3 & 0xff] ^ _rk[20];
    t1 = Te0[s1 >> 24] ^ Te1[(s2 >> 16) & 0xff] ^ Te2[(s3 >>  8) & 0xff] ^ Te3[s0 & 0xff] ^ _rk[21];
    t2 = Te0[s2 >> 24] ^ Te1[(s3 >> 16) & 0xff] ^ Te2[(s0 >>  8) & 0xff] ^ Te3[s1 & 0xff] ^ _rk[22];
    t3 = Te0[s3 >> 24] ^ Te1[(s0 >> 16) & 0xff] ^ Te2[(s1 >>  8) & 0xff] ^ Te3[s2 & 0xff] ^ _rk[23];
    /* round 6: */
    s0 = Te0[t0 >> 24] ^ Te1[(t1 >> 16) & 0xff] ^ Te2[(t2 >>  8) & 0xff] ^ Te3[t3 & 0xff] ^ _rk[24];
    s1 = Te0[t1 >> 24] ^ Te1[(t2 >> 16) & 0xff] ^ Te2[(t3 >>  8) & 0xff] ^ Te3[t0 & 0xff] ^ _rk[25];
    s2 = Te0[t2 >> 24] ^ Te1[(t3 >> 16) & 0xff] ^ Te2[(t0 >>  8) & 0xff] ^ Te3[t1 & 0xff] ^ _rk[26];
    s3 = Te0[t3 >> 24] ^ Te1[(t0 >> 16) & 0xff] ^ Te2[(t1 >>  8) & 0xff] ^ Te3[t2 & 0xff] ^ _rk[27];
    /* round 7: */
    t0 = Te0[s0 >> 24] ^ Te1[(s1 >> 16) & 0xff] ^ Te2[(s2 >>  8) & 0xff] ^ Te3[s3 & 0xff] ^ _rk[28];
    t1 = Te0[s1 >> 24] ^ Te1[(s2 >> 16) & 0xff] ^ Te2[(s3 >>  8) & 0xff] ^ Te3[s0 & 0xff] ^ _rk[29];
    t2 = Te0[s2 >> 24] ^ Te1[(s3 >> 16) & 0xff] ^ Te2[(s0 >>  8) & 0xff] ^ Te3[s1 & 0xff] ^ _rk[30];
    t3 = Te0[s3 >> 24] ^ Te1[(s0 >> 16) & 0xff] ^ Te2[(s1 >>  8) & 0xff] ^ Te3[s2 & 0xff] ^ _rk[31];
    /* round 8: */
    s0 = Te0[t0 >> 24] ^ Te1[(t1 >> 16) & 0xff] ^ Te2[(t2 >>  8) & 0xff] ^ Te3[t3 & 0xff] ^ _rk[32];
    s1 = Te0[t1 >> 24] ^ Te1[(t2 >> 16) & 0xff] ^ Te2[(t3 >>  8) & 0xff] ^ Te3[t0 & 0xff] ^ _rk[33];
    s2 = Te0[t2 >> 24] ^ Te1[(t3 >> 16) & 0xff] ^ Te2[(t0 >>  8) & 0xff] ^ Te3[t1 & 0xff] ^ _rk[34];
    s3 = Te0[t3 >> 24] ^ Te1[(t0 >> 16) & 0xff] ^ Te2[(t1 >>  8) & 0xff] ^ Te3[t2 & 0xff] ^ _rk[35];
    /* round 9: */
    t0 = Te0[s0 >> 24] ^ Te1[(s1 >> 16) & 0xff] ^ Te2[(s2 >>  8) & 0xff] ^ Te3[s3 & 0xff] ^ _rk[36];
    t1 = Te0[s1 >> 24] ^ Te1[(s2 >> 16) & 0xff] ^ Te2[(s3 >>  8) & 0xff] ^ Te3[s0 & 0xff] ^ _rk[37];
    t2 = Te0[s2 >> 24] ^ Te1[(s3 >> 16) & 0xff] ^ Te2[(s0 >>  8) & 0xff] ^ Te3[s1 & 0xff] ^ _rk[38];
    t3 = Te0[s3 >> 24] ^ Te1[(s0 >> 16) & 0xff] ^ Te2[(s1 >>  8) & 0xff] ^ Te3[s2 & 0xff] ^ _rk[39];
    if (Nr > 10) {
	// round 10
        s0 = Te0[t0 >> 24] ^ Te1[(t1 >> 16) & 0xff] ^ Te2[(t2 >>  8) & 0xff] ^ Te3[t3 & 0xff] ^ _rk[40];
        s1 = Te0[t1 >> 24] ^ Te1[(t2 >> 16) & 0xff] ^ Te2[(t3 >>  8) & 0xff] ^ Te3[t0 & 0xff] ^ _rk[41];
        s2 = Te0[t2 >> 24] ^ Te1[(t3 >> 16) & 0xff] ^ Te2[(t0 >>  8) & 0xff] ^ Te3[t1 & 0xff] ^ _rk[42];
        s3 = Te0[t3 >> 24] ^ Te1[(t0 >> 16) & 0xff] ^ Te2[(t1 >>  8) & 0xff] ^ Te3[t2 & 0xff] ^ _rk[43];
	// round 11
        t0 = Te0[s0 >> 24] ^ Te1[(s1 >> 16) & 0xff] ^ Te2[(s2 >>  8) & 0xff] ^ Te3[s3 & 0xff] ^ _rk[44];
        t1 = Te0[s1 >> 24] ^ Te1[(s2 >> 16) & 0xff] ^ Te2[(s3 >>  8) & 0xff] ^ Te3[s0 & 0xff] ^ _rk[45];
        t2 = Te0[s2 >> 24] ^ Te1[(s3 >> 16) & 0xff] ^ Te2[(s0 >>  8) & 0xff] ^ Te3[s1 & 0xff] ^ _rk[46];
        t3 = Te0[s3 >> 24] ^ Te1[(s0 >> 16) & 0xff] ^ Te2[(s1 >>  8) & 0xff] ^ Te3[s2 & 0xff] ^ _rk[47];
        if (Nr > 12) {
	    // round 12
            s0 = Te0[t0 >> 24] ^ Te1[(t1 >> 16) & 0xff] ^ Te2[(t2 >>  8) & 0xff] ^ Te3[t3 & 0xff] ^ _rk[48];
            s1 = Te0[t1 >> 24] ^ Te1[(t2 >> 16) & 0xff] ^ Te2[(t3 >>  8) & 0xff] ^ Te3[t0 & 0xff] ^ _rk[49];
            s2 = Te0[t2 >> 24] ^ Te1[(t3 >> 16) & 0xff] ^ Te2[(t0 >>  8) & 0xff] ^ Te3[t1 & 0xff] ^ _rk[50];
            s3 = Te0[t3 >> 24] ^ Te1[(t0 >> 16) & 0xff] ^ Te2[(t1 >>  8) & 0xff] ^ Te3[t2 & 0xff] ^ _rk[51];
	    // round 13
            t0 = Te0[s0 >> 24] ^ Te1[(s1 >> 16) & 0xff] ^ Te2[(s2 >>  8) & 0xff] ^ Te3[s3 & 0xff] ^ _rk[52];
            t1 = Te0[s1 >> 24] ^ Te1[(s2 >> 16) & 0xff] ^ Te2[(s3 >>  8) & 0xff] ^ Te3[s0 & 0xff] ^ _rk[53];
            t2 = Te0[s2 >> 24] ^ Te1[(s3 >> 16) & 0xff] ^ Te2[(s0 >>  8) & 0xff] ^ Te3[s1 & 0xff] ^ _rk[54];
            t3 = Te0[s3 >> 24] ^ Te1[(s0 >> 16) & 0xff] ^ Te2[(s1 >>  8) & 0xff] ^ Te3[s2 & 0xff] ^ _rk[55];
        }
    }
    /*
     * apply last round and
     * map cipher state to byte array block:
     */
    _rk += (Nr << 2);

    state[0].i = REV_ENDIAN(t0);
    state[1].i = REV_ENDIAN(t1);
    state[2].i = REV_ENDIAN(t2);
    state[3].i = REV_ENDIAN(t3);
    TransposeSelf((uchar*)state);
    SubBytes((uchar*)state, sbox);
    ShiftRows((uchar*)state);
    AddRoundKey((uchar*)state, (uchar*)(_rk));
    TransposeSelf((uchar*)state);
    ct[offset + 0] = REV_ENDIAN(state[0].i);
    ct[offset + 1] = REV_ENDIAN(state[1].i);
    ct[offset + 2] = REV_ENDIAN(state[2].i);
    ct[offset + 3] = REV_ENDIAN(state[3].i);

    iter++;
    goto BEGIN;
}

