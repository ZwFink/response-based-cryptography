#include "hip/hip_runtime.h"
#include "util.cu"
//#define TTABLE 256
//#define USE_SMEM
union u32_t {
    uint i;
    uchar c[4];
};

#if TTABLE == 256
#include "tabs/AES_256.tab"
#define TE(tab, offset, state) (_te[(tab)][((offset) << 8) + (state)])

#elif TTABLE == 128
#include "tabs/AES_128.tab"
#define TE_128_LH(tab, offset, state) (_te[(tab)][((offset) << 8) + (state >> 1)      ] >> ((state & 0x1) << 2))
#define TE_128_HH(tab, offset, state) (_te[(tab)][((offset) << 8) + (state >> 1) + 128] >> ((state & 0x1) << 2))
#define TE(tab, offset, state) ((TE_128_LH(tab, offset, state) & 0x0f) | (TE_128_HH(tab, offset, state) << 4))

#elif TTABLE == 64
#include "tabs/AES_64.tab"
#define TE_64_0(tab, offset, state) (_te[(tab)][((offset) << 8) + (state >> 2)      ] >> ((state & 0x3) << 1))
#define TE_64_1(tab, offset, state) (_te[(tab)][((offset) << 8) + (state >> 2) +  64] >> ((state & 0x3) << 1))
#define TE_64_2(tab, offset, state) (_te[(tab)][((offset) << 8) + (state >> 2) + 128] >> ((state & 0x3) << 1))
#define TE_64_3(tab, offset, state) (_te[(tab)][((offset) << 8) + (state >> 2) + 192] >> ((state & 0x3) << 1))
#define TE(tab, offset, state) ((TE_64_0(tab, offset, state) & 0x03)	\
				| ((TE_64_1(tab, offset, state) & 0x03) << 2) \
				| ((TE_64_2(tab, offset, state) & 0x03) << 4) \
				| ((TE_64_3(tab, offset, state) & 0x03) << 6))

#elif TTABLE == 32
#include "tabs/AES_32.tab"
#define TE_32_0(tab, offset, state) (_te[(tab)][((offset) << 8) + (state >> 3)      ] >> (state & 0x7))
#define TE_32_1(tab, offset, state) (_te[(tab)][((offset) << 8) + (state >> 3) +  32] >> (state & 0x7))
#define TE_32_2(tab, offset, state) (_te[(tab)][((offset) << 8) + (state >> 3) +  64] >> (state & 0x7))
#define TE_32_3(tab, offset, state) (_te[(tab)][((offset) << 8) + (state >> 3) +  96] >> (state & 0x7))
#define TE_32_4(tab, offset, state) (_te[(tab)][((offset) << 8) + (state >> 3) + 128] >> (state & 0x7))
#define TE_32_5(tab, offset, state) (_te[(tab)][((offset) << 8) + (state >> 3) + 160] >> (state & 0x7))
#define TE_32_6(tab, offset, state) (_te[(tab)][((offset) << 8) + (state >> 3) + 192] >> (state & 0x7))
#define TE_32_7(tab, offset, state) (_te[(tab)][((offset) << 8) + (state >> 3) + 224] >> (state & 0x7))
#define TE(tab, offset, state) ((TE_32_0(tab, offset, state) & 0x01)	      \
				| ((TE_32_1(tab, offset, state) & 0x01) << 1) \
				| ((TE_32_2(tab, offset, state) & 0x01) << 2) \
				| ((TE_32_3(tab, offset, state) & 0x01) << 3) \
				| ((TE_32_4(tab, offset, state) & 0x01) << 4) \
				| ((TE_32_5(tab, offset, state) & 0x01) << 5) \
				| ((TE_32_6(tab, offset, state) & 0x01) << 6) \
				| ((TE_32_7(tab, offset, state) & 0x01) << 7))		   
#endif


#define LOAD_U8_G0(s, offset) {					\
	m[0].c[offset] = TE(0, offset, s[0].c[3]);			\
	m[1].c[offset] = TE(1, offset, s[1].c[2]);			\
	m[2].c[offset] = TE(2, offset, s[2].c[1]);			\
	m[3].c[offset] = TE(3, offset, s[3].c[0]);			\
    }

#define LOAD_U8_G1(s, offset) {						\
	m[0].c[offset] = TE(0, offset, s[1].c[3]);			\
	m[1].c[offset] = TE(1, offset, s[2].c[2]);			\
	m[2].c[offset] = TE(2, offset, s[3].c[1]);			\
	m[3].c[offset] = TE(3, offset, s[0].c[0]);			\
    }
#define LOAD_U8_G2(s, offset) {						\
	m[0].c[offset] = TE(0, offset, s[2].c[3]);			\
	m[1].c[offset] = TE(1, offset, s[3].c[2]);			\
	m[2].c[offset] = TE(2, offset, s[0].c[1]);			\
	m[3].c[offset] = TE(3, offset, s[1].c[0]);			\
    }
#define LOAD_U8_G3(s, offset) {						\
	m[0].c[offset] = TE(0, offset, s[3].c[3]);			\
	m[1].c[offset] = TE(1, offset, s[0].c[2]);			\
	m[2].c[offset] = TE(2, offset, s[1].c[1]);			\
	m[3].c[offset] = TE(3, offset, s[2].c[0]);			\
    }

#define SROUND(t, s, r) {						\
	int b = r << 2;							\
	u32_t m[4];							\
	LOAD_U8_G0(s, 0); LOAD_U8_G0(s, 1); LOAD_U8_G0(s, 2); LOAD_U8_G0(s, 3); \
	t[0].i = m[0].i ^ m[1].i ^ m[2].i ^ m[3].i ^ _rk[b + 0].i;	\
	LOAD_U8_G1(s, 0); LOAD_U8_G1(s, 1); LOAD_U8_G1(s, 2); LOAD_U8_G1(s, 3);	\
	t[1].i = m[0].i ^ m[1].i ^ m[2].i ^ m[3].i ^ _rk[b + 1].i;	\
	LOAD_U8_G2(s, 0); LOAD_U8_G2(s, 1); LOAD_U8_G2(s, 2); LOAD_U8_G2(s, 3);	\
	t[2].i = m[0].i ^ m[1].i ^ m[2].i ^ m[3].i ^ _rk[b + 2].i;	\
	LOAD_U8_G3(s, 0); LOAD_U8_G3(s, 1); LOAD_U8_G3(s, 2); LOAD_U8_G3(s, 3);	\
	t[3].i = m[0].i ^ m[1].i ^ m[2].i ^ m[3].i ^ _rk[b + 3].i;	\
    }

__global__ void AES_encrypt(const uint *pt, uint *ct, uint *rek, uint Nr, uint size) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    u32_t *_pt = (u32_t *)pt, *_ct = (u32_t *)ct, *_rk = (u32_t *)rek;
    u32_t _s[4], _t[4];
#ifdef USE_SMEM
    __shared__ uchar sTe0[1024], sTe1[1024], sTe2[1024], sTe3[1024];
    uchar* _te[] = {sTe0, sTe1, sTe2, sTe3};
#if TTABLE == 256
    load_smem(sTe0, dTe0_256, sTe1, dTe1_256, sTe2, dTe2_256, sTe3, dTe3_256);
#elif TTABLE == 128
    load_smem(sTe0, dTe0_128, sTe1, dTe1_128, sTe2, dTe2_128, sTe3, dTe3_128);
#elif TTABLE == 64
    load_smem(sTe0, dTe0_64, sTe1, dTe1_64, sTe2, dTe2_64, sTe3, dTe3_64);
#elif TTABLE == 32
    load_smem(sTe0, dTe0_32, sTe1, dTe1_32, sTe2, dTe2_32, sTe3, dTe3_32);
#endif // TTABLE
#else
#if TTABLE == 256
    uchar* _te[] = {dTe0_256, dTe1_256, dTe2_256, dTe3_256};
#elif TTABLE == 128
    uchar* _te[] = {dTe0_128, dTe1_128, dTe2_128, dTe3_128};
#elif TTABLE == 64
    uchar* _te[] = {dTe0_64, dTe1_64, dTe2_64, dTe3_64};
#elif TTABLE == 32
    uchar* _te[] = {dTe0_32, dTe1_32, dTe2_32, dTe3_32};
#endif // TTABLE
#endif // USE_SMEM

    int iter = 0;

 BEGIN:
    int offset = (iter * NUM_THREADS * NUM_BLOCKS + tid) << 2;
    if (offset >= size) return;

    /*
     * map byte array block to cipher state
     * and add initial round key:
     */
    _rk = (u32_t *)rek;
    _s[0].i = _pt[offset + 0].i ^ _rk[0].i;
    _s[1].i = _pt[offset + 1].i ^ _rk[1].i;
    _s[2].i = _pt[offset + 2].i ^ _rk[2].i;
    _s[3].i = _pt[offset + 3].i ^ _rk[3].i;

    SROUND(_t, _s, 1);
    SROUND(_s, _t, 2);
    SROUND(_t, _s, 3);
    SROUND(_s, _t, 4);
    SROUND(_t, _s, 5);
    SROUND(_s, _t, 6);
    SROUND(_t, _s, 7);
    SROUND(_s, _t, 8);
    SROUND(_t, _s, 9);
    if (Nr > 10) {
	SROUND(_s, _t, 10);
	SROUND(_t, _s, 11);
        if (Nr > 12) {
	    SROUND(_s, _t, 12);
	    SROUND(_t, _s, 13);
        }
    }
    /*
     * apply last round and
     * map cipher state to byte array block:
     */
    _rk += (Nr << 2);
    u32_t m;
    m.c[0] = TE(1, 0, _t[3].c[0]);
    m.c[1] = TE(0, 1, _t[2].c[1]);
    m.c[2] = TE(3, 2, _t[1].c[2]);
    m.c[3] = TE(2, 3, _t[0].c[3]);
    _ct[offset + 0].i = m.i ^ _rk[0].i;
    m.c[0] = TE(1, 0, _t[0].c[0]);
    m.c[1] = TE(0, 1, _t[3].c[1]);
    m.c[2] = TE(3, 2, _t[2].c[2]);
    m.c[3] = TE(2, 3, _t[1].c[3]);
    _ct[offset + 1].i = m.i ^ _rk[1].i;
    m.c[0] = TE(1, 0, _t[1].c[0]);
    m.c[1] = TE(0, 1, _t[0].c[1]);
    m.c[2] = TE(3, 2, _t[3].c[2]);
    m.c[3] = TE(2, 3, _t[2].c[3]);
    _ct[offset + 2].i = m.i ^ _rk[2].i;
    m.c[0] = TE(1, 0, _t[2].c[0]);
    m.c[1] = TE(0, 1, _t[1].c[1]);
    m.c[2] = TE(3, 2, _t[0].c[2]);
    m.c[3] = TE(2, 3, _t[3].c[3]);
    _ct[offset + 3].i = m.i ^ _rk[3].i;

    iter++;
    goto BEGIN;
}

